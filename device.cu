#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount); // �������� ���������� CUDA-����������� ���������

    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found." << std::endl;
        return 1;
    }

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop; // ������������� ��������� hipDeviceProp_t
        hipGetDeviceProperties(&prop, i); // �������� �������� ��� ���������� 'i'

        std::cout << "--- Device Number: " << i << " ---" << std::endl;
        std::cout << "  Device Name: " << prop.name << std::endl;
        std::cout << "  Total Global Memory (bytes): " << prop.totalGlobalMem << std::endl;
        std::cout << "  Shared Memory per Block (bytes): " << prop.sharedMemPerBlock << std::endl;

        std::cout << "  Compute Capability: " << prop.major << "." << prop.minor << std::endl;
        std::cout << "  Max Threads per Block: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "  Multiprocessor Count: " << prop.multiProcessorCount << std::endl;
        std::cout << "  Clock Rate (kHz): " << prop.clockRate << std::endl;
        std::cout << "  Warp Size: " << prop.warpSize << std::endl;
        std::cout << "  Registers Per Block: " << prop.regsPerBlock << std::endl; // ���������
        std::cout << "  Memory Pitch (bytes): " << prop.memPitch << std::endl; // ���������
        std::cout << "  ECC Enabled: " << (prop.ECCEnabled ? "Yes" : "No") << std::endl;
        std::cout << std::endl;
    }

    return 0;
}